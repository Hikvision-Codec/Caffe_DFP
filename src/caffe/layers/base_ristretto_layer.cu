#include "hip/hip_runtime.h"
#include "caffe/layers/base_ristretto_layer.hpp"

#include <math.h>
#include <iostream>
namespace caffe {
template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerOutputs_gpu(Dtype* data,
      const int count) {
Trim2FixedPoint_simulation_gpu(data, count, bw_layer_out_, fl_layer_out_);
}

template <typename Dtype>
__global__ void Trim2FixedPoint_simulation_kernel(Dtype* data, const int cnt,
      const int bit_width, const int fl, Dtype max_data, Dtype min_data) {
	CUDA_KERNEL_LOOP(index, cnt) {
    // Saturate data

    data[index] = fmax(fmin(data[index], max_data), min_data);
    // Round data
    data[index] /= powf(2, -fl);
    data[index] = rint(data[index]);
    data[index] *= powf(2, -fl);
	}
}


template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2FixedPoint_simulation_gpu(Dtype* data, const int cnt,
      const int bit_width, int fl) {
  Dtype max_data = (powf(2, bit_width - 1) - 1) * powf(2, -fl);
  Dtype min_data = -powf(2, bit_width - 1) * powf(2, -fl);
  Trim2FixedPoint_simulation_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bit_width, fl, max_data, min_data);
}

// Explicit instantiations
template void BaseRistrettoLayer<double>::QuantizeLayerOutputs_gpu(
    double* top_data, const int top_count);
template void BaseRistrettoLayer<float>::QuantizeLayerOutputs_gpu(
    float* top_data, const int top_count);
template void BaseRistrettoLayer<double>::Trim2FixedPoint_simulation_gpu(double* data,
    const int cnt, const int bit_width, int fl);
template void BaseRistrettoLayer<float>::Trim2FixedPoint_simulation_gpu(float* data,
    const int cnt, const int bit_width, int fl);
}  // namespace caffe


