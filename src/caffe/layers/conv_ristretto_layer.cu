#include <vector>
#include <hip/hip_runtime.h>

#include "caffe/util/im2col.hpp"
#include "caffe/layers/conv_ristretto_layer.hpp"

namespace caffe {

template <typename Dtype>
void ConvolutionRistrettoLayer<Dtype>::Forward_gpu(
      const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {   
  // Do forward propagation
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }
    }
    // Trim layer output
    if (this->quanti_out_ == true) { 
      this->QuantizeLayerOutputs_gpu(top_data, top[i]->count());
    }
  }
}

template <typename Dtype>
void ConvolutionRistrettoLayer<Dtype>::Backward_gpu(
      const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
      const vector<Blob<Dtype>*>& bottom) {
 //NOT IMPLEMENT
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionRistrettoLayer);

}  // namespace caffe
